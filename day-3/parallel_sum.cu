
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void parallel_reduction(int *input, int *output, int N)
    {
        __shared__ int temp[8];
        int tid = threadIdx.x;

        temp[tid] = input[tid];
        __syncthreads();

            // Step 2: Perform reduction in shared memory
        for (int stride = 1; stride < N; stride *= 2) {
            if (tid % (2 * stride) == 0 && (tid + stride) < N) {
                temp[tid] += temp[tid + stride];  // Change to max/min/product if needed
            }
            __syncthreads();  // Sync after each stride to avoid race conditions ( Simply that every thread is on same plcae)
        }

        // Step 3: Thread 0 writes the final result
        if (tid == 0) {
            output[0] = temp[0];
        }
        
    }


int main() {
    const int N = 8;
    int h_input[N] = {1, 2, 3, 4, 5, 6, 7, 8};  // Input array
    int h_output = 0;

    int *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, sizeof(int));

    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 1 block of N threads
    parallel_reduction<<<1, N>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(&h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);
    printf("Sum = %d\n", h_output);  // Expected: 36

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
